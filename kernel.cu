#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include "cudaHeader.cuh"



void construct(keyEntry* permutation, keyEntry* matrix, keyEntry* baseMatrix, matrixIndexPair * constructionGuide, const int permutation_size, const int matrix_size, const int rows)
{
	device_Construct << < rows, matrix_size >> > (permutation, matrix, baseMatrix, constructionGuide, permutation_size);
}

__global__ void device_Construct(keyEntry* permutation, keyEntry* matrix, keyEntry* baseMatrix, matrixIndexPair* ConstructionGuide, const int permutation_size)
{
	keyEntry fIndex = blockIdx.x * (permutation_size) + ConstructionGuide[threadIdx.x].lowIndex;
	keyEntry sIndex = blockIdx.x * (permutation_size) + ConstructionGuide[threadIdx.x].highIndex;

	keyEntry fVal = permutation[fIndex];
	keyEntry sVal = permutation[sIndex];


	unsigned short row, col;

	// ensure that column always gets the higher number and row gets the lower number
	// the ternary operators are slight performance improvement over an if-else
	// I think this is because they are able to compile to max-load instructions which don't result in divergence
	row = min(fVal, sVal);
	col = max(fVal, sVal);

	row--;
	col--;

	int	outputIndex = col - 1 + (row * (permutation_size - 2) - ((row - 1) * row) / 2);
	keyEntry valM = ConstructionGuide[threadIdx.x].highIndex - ConstructionGuide[threadIdx.x].lowIndex;
	keyEntry valB = baseMatrix[outputIndex];
	matrix[blockIdx.x * blockDim.x + outputIndex] = abs(valM-valB);
}

void summation(keyEntry* matrix, keyEntry* rowSums, int* incGuide, const int reduction_size, const int matrix_size, const int reductions, const int rows, const int threads)
{
	device_summation << < rows, threads, reduction_size * sizeof(keyEntry) >>> (matrix, rowSums, incGuide, reduction_size, matrix_size, reductions);
}

__global__ void device_summation(keyEntry* matrix, keyEntry* rowSums, int* incGuide, const int reduction_size, const int matrix_size, const int reductions)
{
	extern __shared__ keyEntry reductionData[];

	//transfer from global to shared memory in two halves
	//for the second half, pad with zeroes where neccessary.
	reductionData[threadIdx.x] = matrix[blockIdx.x * (matrix_size)+threadIdx.x];
	reductionData[threadIdx.x + reduction_size / 2] = (matrix_size-1 < threadIdx.x + reduction_size / 2) ? 0 : matrix[blockIdx.x * (matrix_size)+threadIdx.x + reduction_size / 2];

	__syncthreads();

	for (int i = 0; i < reductions; i++)
	{
		if (threadIdx.x < incGuide[i]) reductionData[threadIdx.x] += reductionData[threadIdx.x + incGuide[i]];
		__syncthreads();
	}

	if (threadIdx.x == 0) rowSums[blockIdx.x] = reductionData[0];
	
}

void maxima(keyEntry* rowSums, keyEntry* gpu_max, int* guide, const int rows, const int reduction_size, const int reductions, int threads)
{
	device_maxima << < 1, threads, reduction_size * sizeof(keyEntry) >> > (rowSums, gpu_max, guide, reduction_size, reductions, rows);
}

__global__ void device_maxima( keyEntry* rowSums, keyEntry* gpu_max, int* incGuide, const int reduction_size, const int reductions, const int rows)
{
	extern __shared__ keyEntry reductionData[];

	reductionData[threadIdx.x] = rowSums[threadIdx.x];
	reductionData[threadIdx.x + reduction_size / 2] = (rows - 1 < threadIdx.x + reduction_size / 2) ? 0 : rowSums[threadIdx.x + reduction_size / 2];

	__syncthreads();

	for (int i = 0; i < reductions; i++)
	{
		if (threadIdx.x < incGuide[i]) reductionData[threadIdx.x] = max(reductionData[threadIdx.x],reductionData[threadIdx.x + incGuide[i]]);
		__syncthreads();
	}

	if (threadIdx.x == 0) *gpu_max = max(*gpu_max,reductionData[0]);
}